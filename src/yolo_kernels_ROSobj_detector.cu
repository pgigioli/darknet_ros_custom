#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"

#include "ROS_interface.h"

extern "C" {
#include "network.h"
#include "detection_layer.h"
#include "cost_layer.h"
#include "utils.h"
#include "parser.h"
#include "box.h"
#include "image.h"
#include <sys/time.h>
}

#ifdef OPENCV
#include "opencv2/highgui/highgui.hpp"
#include "opencv2/imgproc/imgproc.hpp"
extern "C" image ipl_to_image(IplImage* src);
extern "C" void convert_yolo_detections(float *predictions, int classes, int num, int square, int side, int w, int h, float thresh, float **probs, box *boxes, int only_objectness);
extern "C" void draw_yolo(image im, int num, float thresh, box *boxes, float **probs);

extern "C" char *voc_names[];
extern "C" image voc_labels[];

static float **probs;
static box *boxes;
static network net;
static image in;
static image in_s;
static image det_s;
static float fps = 0;
static float demo_thresh = 0;

static ROS_box *ROI_boxes;

void fetch_in_thread()
{
    cv::Mat cv_frame = get_Mat_image();
    IplImage ROS_img = cv_frame;
    in = ipl_to_image(&ROS_img);
    rgbgr_image(in);
    in_s = resize_image(in, net.w, net.h);
    free_image(in);
    return;
}

void detect_in_thread()
{
    float nms = .4;

    detection_layer l = net.layers[net.n-1];
    float *X = det_s.data;
    float *predictions = network_predict(net, X);
    free_image(det_s);
    convert_yolo_detections(predictions, l.classes, l.n, l.sqrt, l.side, 1, 1, demo_thresh, probs, boxes, 0);
    if (nms > 0) do_nms(boxes, probs, l.side*l.side*l.n, l.classes, nms);
    printf("\033[2J");
    printf("\033[1;1H");
    printf("\nFPS:%.0f\n",fps);

    // extract the bounding boxes and send them to ROS
    int total = l.side*l.side*l.n;
    int i, j;
    int count = 0;
    for(i = 0; i < total; ++i){
        float xmin = boxes[i].x - boxes[i].w/2.;
        float xmax = boxes[i].x + boxes[i].w/2.;
        float ymin = boxes[i].y - boxes[i].h/2.;
        float ymax = boxes[i].y + boxes[i].h/2.;

        if (xmin < 0) xmin = 0;
        if (ymin < 0) ymin = 0;
        if (xmax > 1) xmax = 1;
        if (ymax > 1) ymax = 1;

	// iterate through possible boxes and collect the bounding boxes
        for(j = 0; j < l.classes; ++j){
            if (probs[i][j]) {
		float x_center = (xmin+xmax)/2;
		float y_center = (ymin+ymax)/2;
		float bbox_width = xmax - xmin;
		float bbox_height = ymax - ymin;

		// define bounding box 
		// bbox must be 1% size of frame (3.2x2.4 pixels)
		if (bbox_width > 0.01 && bbox_height > 0.01) {
   		     ROI_boxes[count].x = x_center;
                     ROI_boxes[count].y = y_center;
                     ROI_boxes[count].w = bbox_width;
                     ROI_boxes[count].h = bbox_height;
                     ROI_boxes[count].Class = j;
		   count++;
		}
		
		//printf("%f %f\n", x_center*320, y_center*240);
	    }
        }
    }
    
    // create array to store found bounding boxes
    // if no object detected, make sure that ROS knows that num = 0
    if (count == 0) {
        ROI_boxes[0].num = 0;
    } else {
        ROI_boxes[0].num = count;
    }

    return;
}

extern "C" void load_network(char *cfgfile, char *weightfile, float thresh, int cam_index)
{
    demo_thresh = thresh;
    printf("YOLO demo\n");
    net = parse_network_cfg(cfgfile);
    if(weightfile){
        load_weights(&net, weightfile);
    }
    set_batch_network(&net, 1);

    srand(2222222);

    detection_layer l = net.layers[net.n-1];
    int j;

    boxes = (box *)calloc(l.side*l.side*l.n, sizeof(box));
    ROI_boxes = (ROS_box *)calloc(l.side*l.side*l.n, sizeof(ROS_box));
    probs = (float **)calloc(l.side*l.side*l.n, sizeof(float *));
    for(j = 0; j < l.side*l.side*l.n; ++j) probs[j] = (float *)calloc(l.classes, sizeof(float *));
}

extern "C" ROS_box *demo_yolo()
{
    struct timeval tval_before, tval_after, tval_result;
    gettimeofday(&tval_before, NULL);
    fetch_in_thread();
    detect_in_thread();
    det_s = in_s;

    gettimeofday(&tval_after, NULL);
    timersub(&tval_after, &tval_before, &tval_result);
    float curr = 1000000.f/((long int)tval_result.tv_usec);
    fps = .9*fps + .1*curr;
    return ROI_boxes;
}
#else
extern "C" void demo_yolo(char *cfgfile, char *weightfile, float thresh, int cam_index){
    fprintf(stderr, "YOLO demo needs OpenCV for webcam images.\n");
}
#endif

